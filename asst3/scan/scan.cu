#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}


__global__ void 
upsweep_kernel(int total_threads, int N, int two_dplus1, int* result) {
    long long offset = blockIdx.x * blockDim.x * two_dplus1;
    long long i = offset + threadIdx.x * two_dplus1;

    if (i < N)
        result[i + two_dplus1 - 1] += result[i + two_dplus1 / 2 - 1];
}

__global__ void
downsweep_kernel(int total_threads, int N, int two_dplus1, int* result) {
    long long offset = blockIdx.x * blockDim.x * two_dplus1;
    long long i = offset + threadIdx.x * two_dplus1;

    if (i < N) {
        int t = result[i + two_dplus1 / 2 - 1];
        result[i + two_dplus1 / 2 - 1] = result[i + two_dplus1 - 1];
        result[i + two_dplus1 - 1] += t;
    }
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{
    for (int two_d = 1; two_d <= N / 2; two_d *= 2) {
        int two_dplus1 = 2 * two_d;
        int total_threads = N / (2 * two_d);
        int blocks = (total_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        upsweep_kernel<<<blocks, THREADS_PER_BLOCK>>>(total_threads, N, two_dplus1, result);
    }

    hipMemset(&result[N - 1], 0, sizeof(int));

    for (int two_d = N / 2; two_d >= 1; two_d /= 2) {
        int two_dplus1 = 2 * two_d;
        int total_threads = N / (2 * two_d);
        int blocks = (total_threads + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
        downsweep_kernel<<<blocks, THREADS_PER_BLOCK>>>(total_threads, N, two_dplus1, result);
    }
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    printf("calling with %d\n", rounded_length);
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();
    exclusive_scan(device_input, rounded_length, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    // printf("\n---input---\n");
    // for (int i = 0; i < N; i++) {
    //     printf("%d, ", inarray[i]);
    // }
    // printf("\n---input---\n");

    // printf("\n---output---\n");
    // for (int i = 0; i < N; i++) {
    //     printf("%d, ", resultarray[i]);
    // }
    // printf("\n---output---\n");

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


__global__ void
predicate_kernel(int* device_input, int* predicate, int N) {
    long long tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    if (tid == N - 1) {
        predicate[tid] = 0;
    } else {
        predicate[tid] = (device_input[tid] == device_input[tid + 1]);
    }

    // printf("predicate[%d] = %d\n", tid, predicate[tid]);
    
}

__global__ void
map_index_kernel(int* prefix_sum_array, int* output, int length) {
    long long tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= length) return;

    if (prefix_sum_array[tid] != prefix_sum_array[tid + 1])
        output[prefix_sum_array[tid]] = tid;
        
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {
    long long rounded_length = nextPow2(length);

    int *predicate;
    hipMalloc((void **)&predicate, rounded_length * sizeof(int));
    int blocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    predicate_kernel<<<blocks, THREADS_PER_BLOCK>>>(device_input, predicate, length);


    int *result;
    hipMalloc((void **)&result, rounded_length * sizeof(int));
    hipMemcpy(result, predicate, rounded_length * sizeof(int), hipMemcpyDeviceToDevice);
    exclusive_scan(predicate, rounded_length, result);

    // for (int i = 0; i < length; i++) {
    //     printf("%d, ", h_res[i]);
    // }

    int output_size;
    hipMemcpy(&output_size, &result[length - 1], sizeof(int), hipMemcpyDeviceToHost);
    // printf("--%d--\n", output_size);

    map_index_kernel<<<blocks, THREADS_PER_BLOCK>>>(result, device_output, length);

    hipFree(predicate);
    hipFree(result);
    return output_size; 
}

//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    // printf("\n---input---\n");
    // for (int i = 0; i < length; i++) {
    //     printf("%d, ", input[i]);
    // }
    // printf("\n---input---\n");

    // printf("\n---output---\n");
    // for (int i = 0; i < length; i++) {
    //     printf("%d, ", output[i]);
    // }
    // printf("\n---output---\n");

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
